#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>


/*********************************/
/** constants/define statements **/
/*********************************/

#define THREADS_PER_BLOCK 1024
#define MAX_BLOCKS 65535
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define BUFFER_CHAR 'A'
#define PAD_AMT 5


/**********************/
/** function headers **/
/**********************/

void usage(void);
int init_data(char** data, unsigned int num_chars);
int init_data_pad(char** data, unsigned int num_chars);
int read_data(char* data, char* file, unsigned int num_genomes,
                unsigned int genome_len, unsigned int buffer_len,
                unsigned int buffed_len, unsigned int vicinity);
void free_data(char* data, unsigned int num_genomes);
__global__ void readcmp(char* a, char* b, char* result,
                            unsigned long nthreads, unsigned int str_len, 
		                    unsigned int vicinity, unsigned int tstride);
__global__ void reduce(char *g_idata, char *g_odata, unsigned long nthreads, 
                            unsigned int str_len, unsigned int pop_thresh);
void print_device_info(void);
unsigned int next_power_2(unsigned int v);
unsigned int log_2(unsigned int v);


/***************/
/** functions **/
/***************/

/*
 * Function - gpuAssert
 *
 * Inputs:
 *      code - gpu error code
 *      file - current source file 
 *      line - line within this file
 *      abort - if true, the prgram aborts
 *
 * Description:
 *      This function checks the cuda error code, and aborts if it is not a
 *      success.
 */

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n",hipGetErrorString(code),file,line);
      if (abort) exit(code);
   }
}


/*
 * Kernel - readcmp
 *
 * Arguments:
 *      a - a pointer to one base
 *      b - a pointer to the other base
 *
 * Description:
 *      TODO...
 */

__global__ void readcmp(char *a, char *b, char *result, unsigned long nthreads, unsigned int str_len, unsigned int vicinity, unsigned int tstride) {

	extern __shared__ char sdata[];
	// Find index of this thread
	unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned long tid = x + y * blockDim.x * gridDim.x;
	unsigned long i = threadIdx.x; //local block tid

    int j, k;

	while(tid < nthreads) {
		//result[tid] = a[tid+PAD_AMT] ^ b[tid+PAD_AMT];
		result[tid] = a[tid+PAD_AMT];
		/*__syncthreads();

        if (result[tid]!=0) {
		    for (j=1; j<=vicinity; j++) {
			    if(result[tid+j]!=0)
                    break;
            }
            for (k=1; k<j; k++) {
                result[tid+k]=0xff;
            }
		}*/

		//sdata[i] = a[tid+PAD_AMT] ^ b[tid+PAD_AMT-1];
        //__syncthreads();

        /*if (sdata[i] != 0) {
            for (j=1; j<=vicinity; j++) {
                if(sdata[i+j]!=0)
                    break;
            }
            for (k=1; k<j; k++) {
                sdata[i+k]=0xff;
            }
        }*/

        /*result[tid] &= sdata[i];
        __syncthreads();
		sdata[i] = b[tid+PAD_AMT] ^ a[tid+PAD_AMT-1];
        __syncthreads();*/

        /*if (sdata[i]!=0) {
            for (j=1; j<=vicinity; j++) {
                if(sdata[i+j]!=0)
                    break;
            }
            for (k=1; k<j; k++) {
                sdata[i+k]=0xff;
            }
        }*/

        //result[tid] &= sdata[i];
        //__syncthreads();
    	//result[tid] = (result[tid]==0)?0:1;
		
        tid += tstride;
    }
}

__global__ void reduce(char *g_idata, char *g_odata, unsigned long nthreads, unsigned int str_len, unsigned int pop_thresh) {
	extern __shared__ char sdata[];
	// each thread loads one element from global to shared mem
	unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned long i = x + y * blockDim.x * gridDim.x; //global tid

	unsigned int tid = threadIdx.x; //local block tid

    if(i < nthreads) {
	    sdata[tid] = g_idata[i];
	    __syncthreads();
	    // do reduction in shared mem
	    for(unsigned int s=str_len/2; s>0; s >>= 1){
		    if (tid<s) {
			    sdata[tid] += sdata[tid + s];
		    }
		    __syncthreads();
	    }

	    // write result for this block to global mem
	    if (tid%str_len == 0) {
            g_odata[i/str_len] = (sdata[tid]<=pop_thresh)?1:0;
	    }
    }
}


/*
 * Function - main
 *
 * Arguments:
 *      argc - the number of command line arguments
 *      argv - an array of the command line arguments
 *
 * Outputs:
 *      int - 0 if success, 1 if failure
 *
 * Description:
 *      TODO...
 */

int main(int argc, char *argv[]) {

    /* check the number of command line arguments */
    if(argc != 7) {
        usage();
        return 1;
    }

    /* get arguments */
    char* file_1;
    char* file_2;
    unsigned int num_genomes, genome_len, buffed_len, 
                 buffer_len, vicinity, errors, pop_thresh;

    file_1 = argv[1];
    file_2 = argv[2];    
    num_genomes = atoi(argv[3]);
    genome_len = atoi(argv[4]);
    errors = atoi(argv[5]);
    vicinity = atoi(argv[6]);

    /* calculate important values */
    pop_thresh = (vicinity-1)*(errors-1) + errors; //opocount threshold
    buffed_len = next_power_2(genome_len); //genome length + buffer space
    buffer_len = buffed_len - genome_len; //difference bw genome len and buf len
    unsigned long num_chars = num_genomes*buffed_len;

    /* initialize and allocate strings to compare */
    char* genome_1_data; //first genome data
    char* genome_2_data; //second genome data
    char* result_data;   //xor result data
    char* reduce_data;   //sum of "errors" in each string
    if(init_data_pad(&genome_1_data, num_chars))  return 1;
    if(init_data_pad(&genome_2_data, num_chars))  return 1;
    if(init_data(&result_data, num_chars))  return 1;
    if(init_data(&reduce_data, num_genomes)) return 1;

    /* read in the data */
    if(read_data(genome_1_data, file_1, num_genomes, genome_len, buffer_len, buffed_len, vicinity))  return 1;
    if(read_data(genome_2_data, file_2, num_genomes, genome_len, buffer_len, buffed_len, vicinity))  return 1;

	/* create timing events */
    //clock_t start_c, end_c;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    /* initialize and allocate memoer for GPU input and output arrays */
    char *dev_genome_1_data;
    char *dev_genome_2_data;
    char *dev_result_data;
    char *dev_reduce_data;
	gpuErrchk( hipMalloc((void**)&dev_genome_1_data, (num_chars+PAD_AMT)*sizeof(char)));
	gpuErrchk( hipMalloc((void**)&dev_genome_2_data, (num_chars+PAD_AMT)*sizeof(char)));
	gpuErrchk( hipMalloc((void**)&dev_result_data, num_chars*sizeof(char) ));
	gpuErrchk( hipMalloc((void**)&dev_reduce_data, num_genomes*sizeof(char) ));

	/* set start time */
	hipEventRecord(start, 0);
    //start_c = clock(); //begin timing computation

    /* copy data to GPU */	
	gpuErrchk(hipMemcpy( dev_genome_1_data, genome_1_data, 
        (num_genomes*genome_len+PAD_AMT)*sizeof(char), hipMemcpyHostToDevice ));
	gpuErrchk(hipMemcpy( dev_genome_2_data, genome_2_data, 
        (num_genomes*genome_len+PAD_AMT)*sizeof(char), hipMemcpyHostToDevice ));	

    /* figure out thread count and dimensions for GPU */
    unsigned int num_blocks_x = 256;
    unsigned int num_blocks_y = 1;
    unsigned int threads_per_block = buffed_len;
    unsigned int tstride = threads_per_block*num_blocks_x*num_blocks_y;
    dim3 grid_size(num_blocks_x, num_blocks_y, 1);
    unsigned int log_len = log_2(buffed_len); //TODO: do we need this??

    printf("Number of threads per block: %d\n", threads_per_block);
    printf("Grid Dim X: %d\n", num_blocks_x);
    printf("Grid Dim Y:: %d\n", num_blocks_y);
    printf("num_chars: %lu\n", num_chars);

	/* set start time */
	//hipEventRecord(start,0);
    //start_c = clock(); //begin timing computation


      /******************/
     /** START TIMING **/
    /******************/

    /*========================================================================*/


	/* create and run GPU threads */
	readcmp<<<grid_size,threads_per_block,threads_per_block>>>(dev_genome_1_data, dev_genome_2_data, dev_result_data, num_chars, buffed_len, vicinity, tstride);
    gpuErrchk(hipDeviceSynchronize());
	//reduce<<<grid_size,THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_result_data, dev_reduce_data, num_chars, buffed_len, pop_thresh);
    //gpuErrchk(hipDeviceSynchronize());


    /*========================================================================*/

    /****************/
    /** END TIMING **/
    /****************/


	/* set stop time */
	//hipEventRecord(stop,0);
	//hipEventSynchronize( stop );	
    //end_c = clock(); //end timing computation
	gpuErrchk(hipMemcpy( reduce_data, dev_reduce_data, 
	            num_genomes*sizeof(char), hipMemcpyDeviceToHost ));
	gpuErrchk(hipMemcpy( result_data, dev_result_data,
                num_chars*sizeof(char), hipMemcpyDeviceToHost ));

    printf("\n---------------------------------------------------------------\n");
    for(unsigned int i = 80000256-1024; i < 80000256; i ++) {
        printf("%d: %c %c %d\n", i, genome_1_data[i+5], result_data[i], (genome_1_data[i+5] != result_data[i])?1:0);
    }

	hipEventRecord(stop,0);
	hipEventSynchronize( stop );
    //end_c = clock(); //end timing computation

	/* calculate elapsed time for GPU computation */
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to complete comparison %1.4f ms\n", elapsedTime);
    //printf("Total time of computation: %f\n", (end_c-start_c)/(double)CLOCKS_PER_SEC);
	
	/* print out result_data to check */
	/*for (int i = 0; i < num_chars; i++) {
                printf("%d",*(result_data+i));
                printf("\n");
        }
	printf("\n");*/

    /* print out reduce_data to check */
	//for (int i = 0; i < 2100; i++) {
	//	printf("%d",*(reduce_data+i));
    //    	printf("\n");
    //    }

    /* free and destroy all allocated information */
	hipFree(dev_genome_1_data);
	hipFree(dev_genome_2_data);
	hipFree(dev_result_data);
	hipFree(dev_reduce_data);

	hipEventDestroy(start);
	hipEventDestroy(stop);

    //hipHostFree(genome_1_data);
    //hipHostFree(genome_2_data);
    //hipHostFree(result_data);
    //hipHostFree(reduce_data);
    free(genome_1_data);
    free(genome_2_data);
    free(result_data);
    free(reduce_data);
}

/*
 * Function - usage
 *
 * Description:
 *      Just prints the usage invariant for this program.
 */
void usage(void) {
    printf("\nUsage:\n");
    printf("\t./a.out <file_1> <file_2> <num_genomes> <genome_len> <errors> <vicinity>\n\n");
}


/*
 * Function - init_data
 *
 * Arguments:
 *      data - the array in which to place data
 *      num_chars - the number of chars to allocate
 *
 * Outputs:
 *      int - 0 if success, 1 if failure
 *
 * Description:
 *      This function initializes a data array. Pretty simple to follow.
 */
int init_data(char** data, unsigned int num_chars) {
    
    /* allocate pointers for the genome strings */
    *data = (char*)malloc(num_chars * sizeof(char));
    //hipHostAlloc((void**)data,num_chars*sizeof(char),hipHostMallocDefault);
    if(NULL == *data) {
        printf("init_data - malloc failed\n");
        return 1;
    }

    return 0; //SUCCESS
}


/*
 * Function - init_data_pad TODO
 *
 * Arguments:
 *      data - the array in which to place data
 *      num_chars - the number of chars to allocate
 *
 * Outputs:
 *      int - 0 if success, 1 if failure
 *
 * Description:
 *      This function initializes a data array. Pretty simple to follow.
 */
int init_data_pad(char** data, unsigned int num_chars) {
    
    /* allocate pointers for the genome strings */
    *data = (char*)malloc((num_chars + PAD_AMT) * sizeof(char));
    //hipHostAlloc((void**)data,(num_chars+PAD_AMT)*sizeof(char),hipHostMallocDefault);
    if(NULL == *data) {
        printf("init_data - malloc failed\n");
        return 1;
    }

    return 0; //SUCCESS
}


/*
 * Function - read_data
 *
 * Arguments:
 *      data - the char* to which the data will be written 
 *      file - the filae that contains the genomes we care about
 *      num_genomes - the numebr of genomed to read 
 *      genome_len - the length of the genomes
 *      buffer_len - the length of the buffer at the end of each genome
 *      buffed_len - length of genome + buffer
 *
 * Outputs:
 *      int - 0 if success, 1 if failure
 *
 * Description:
 *      This function reads in all of the genome data from the given genome
 *      file. Each line contains a genome read, and this is read into each
 *      string.
 */
int read_data(char* data, char* file, unsigned int num_genomes, 
                unsigned int genome_len, unsigned int buffer_len,
                unsigned int buffed_len,unsigned int vicinity) {

    /* basic info and variables */
    FILE* ifp; //ifp: "in file pointer"
    char* mode = "r";

    /* open the file */
    ifp = fopen(file, mode);
    if(NULL == ifp) {
        printf("Can't open input file %s!\n", file);
        return 1;
    }

    /* create read in buffer */
    char* buf = (char*)malloc((genome_len+2)*sizeof(char));

    /* initialize the padding at beginning of array */
    for(int i = 0; i < PAD_AMT; i++) {
        *(data + i) = BUFFER_CHAR;
    }
    int limit_len;

    /* calculate the limit to which we will read data */
    if (buffer_len < vicinity)
	    limit_len = buffed_len - vicinity;
    else
        limit_len = genome_len;    

    /* read in the file */
    for(int i = 0; i < num_genomes; i++) {

        //printf("%d\n",i);
        if(NULL != fgets(buf, genome_len + 2, ifp)) {
            
            for(int j = 0; j < limit_len; j++) {
                *(data + PAD_AMT + i*buffed_len + j) = buf[j];
            }

            for(int j = 0; j < buffed_len-limit_len; j++) {
                *(data + PAD_AMT + i*buffed_len + limit_len + j) = BUFFER_CHAR;
            }

            //buf[genome_len] = '\0';
            //printf("%s\n", buf);
        }
        else {
            printf("Failed to read from the file\n");
            return 1;
        }
    }

    /* close the file */
    fclose(ifp);
    free(buf);

    return 0; //SUCCESS
}


/*
 * Function - print_device_info
 *
 * Description:
 *      Prints valuable information out regarding the CUDA-capable devices
 *      in this system.
 */
void print_device_info(void) {
    hipDeviceProp_t  prop;

    int count;
    hipGetDeviceCount( &count ); 
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( " --- General Information for device %d ---\n", i ); 
        printf( "Name: %s\n", prop.name );
        printf( "Compute capability: %d.%d\n", prop.major, prop.minor ); 
        printf( "Clock rate: %d\n", prop.clockRate );
        printf( "Device copy overlap: " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" ); 
        else
            printf( "Disabled\n" );
        printf( "Kernel execition timeout : " ); 
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" ); 
        else
            printf( "Disabled\n" );
        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n", prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n", prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
            prop.maxThreadsDim[0], prop.maxThreadsDim[1],
            prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
            prop.maxGridSize[0], prop.maxGridSize[1],
            prop.maxGridSize[2] );
        printf( "\n" );
    }
}


/*
 * Function - next_power_2
 *
 * Arguments:
 *      v - the value for which we want to find the next power of 2
 *
 * Outputs:
 *      unsigned int - the next power of 2 greater than v
 *
 * Description:
 *      This code basically rounds v up to the next highest power of 2. So if
 *      v was 2, this function would return 2. If v was 15, this function would
 *      return 16. Etcetera.
 *
 *  Source:
 *      http://graphics.stanford.edu/~seander/bithacks.html
 */
unsigned int next_power_2(unsigned int v) {
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

unsigned int log_2(unsigned int v) {
	unsigned int r=0;
	while (v >>= 1) // unroll for more speed...
	{
		r++;
	}
	return r;
}
