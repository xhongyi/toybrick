#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>


/*********************************/
/** constants/define statements **/
/*********************************/

#define THREADS_PER_BLOCK 1024
#define MAX_BLOCKS 65535
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define BUFFER_CHAR 'A'
#define PAD_AMT 5


/**********************/
/** function headers **/
/**********************/

void usage(void);
int init_data(char** data, unsigned int num_chars);
int init_data_pad(char** data, unsigned int num_chars);
int read_data(char* data, char* file, unsigned int num_genomes,
                unsigned int genome_len, unsigned int buffer_len,
                unsigned int buffed_len, unsigned int vicinity);
void free_data(char* data, unsigned int num_genomes);
__global__ void readcmp(char* a, char* b, /*char* result,*/
                            unsigned long nthreads, unsigned int str_len, 
                            unsigned int vicinity, unsigned int tstride, 
                            char *reduce, unsigned int pop_thresh, 
                            unsigned int threads_per_block, 
                            unsigned int shift_amt);
__global__ void reduce(char *g_idata, char *g_odata, unsigned long nthreads, 
                            unsigned int str_len, unsigned int pop_thresh,
                            unsigned int tstride);
void print_device_info(void);
unsigned int next_power_2(unsigned int v);
unsigned int log_2(unsigned int v);


/***************/
/** functions **/
/***************/

/*
 * Function - gpuAssert
 *
 * Inputs:
 *      code - gpu error code
 *      file - current source file 
 *      line - line within this file
 *      abort - if true, the prgram aborts
 *
 * Description:
 *      This function checks the cuda error code, and aborts if it is not a
 *      success.
 */

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n",hipGetErrorString(code),file,line);
      if (abort) exit(code);
   }
}


/*
 * Kernel - readcmp
 *
 * Inputs:
 *      a - a pointer to one read
 *      b - a pointer to the other read
 *      nthreads - the maximum number of threads used
 *      str_len - the length of the reads to be compared (this is a power of 2)
 *      vicinity - the bit-flip vicinity
 *      tstride - the read index stride distance for each thread
 *      pop_thresh - the pop count threshold
 *      thread_per_block - the number of threads per block
 *      shift_amt - the maximum shift amount between the reads
 *
 * Outputs:
 *      reduce - the array returned which contains 0 for "reads match" or 1 for 
 *               "reads don't match"
 *
 * Description:
 *      This Kernel does two things: for each pair of reads, it compares the
 *      characters and stores the bit-op results. It then performas a sum
 *      reduction on each read comparison and checks if it is above the edit
 *      threshold.
 */

__global__ void readcmp(/*char *dev_test_array,*/ char *a, char *b, /*char *result,*/ unsigned long nthreads, unsigned int str_len, unsigned int vicinity, unsigned int tstride, char *reduce, unsigned int pop_thresh, unsigned int threads_per_block, unsigned int shift_amt) {

    // Set up shared memory
    extern __shared__ char shared_data[];
    char* sdata = (char *)shared_data;
    char* result = (char *)&shared_data[threads_per_block];

    // Find index of this thread
    unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned long tid = x + y * blockDim.x * gridDim.x;
    unsigned long i = threadIdx.x; //local block tid

    int j, k;

    while(tid < nthreads) {

        //make the first xor comparison without shifting
        result[i] = a[tid+PAD_AMT] ^ b[tid+PAD_AMT];
        __syncthreads();

        //check the vicinity for 100...01
        if (result[i]!=0) {
            for (j=1; j<vicinity; j++) {
                if(result[i+j]!=0)
                    break;
            }
            if (result[i+j]!=0) {
                for (k=1; k<j; k++) {
                    result[i+k]=0xff;
                }
            }
        }

        //make the remaining xor comparisons with shifting up until the limit
        for(unsigned int cur_shift = 1; cur_shift <= shift_amt; cur_shift++) {
            __syncthreads();
            sdata[i] = a[tid+PAD_AMT] ^ b[tid+PAD_AMT-cur_shift]; //shift b left
            __syncthreads();

            //check vicinity
            if (sdata[i] != 0) {
                for (j=1; j<vicinity; j++) {
                    if(sdata[i+j]!=0)
                        break;
                }
                if (sdata[i+j]!=0) {
                    for (k=1; k<j; k++) {
                        sdata[i+k]=0xff;
                    }
                }
            }

            //AND result
            __syncthreads();
            result[i] = result[i] && sdata[i];

            sdata[i] = b[tid+PAD_AMT] ^ a[tid+PAD_AMT-cur_shift]; //shift a left
            __syncthreads();

            //check vicinity
            if (sdata[i]!=0) {
                for (j=1; j<vicinity; j++) {
                    if(sdata[i+j]!=0)
                        break;
                }
                if (sdata[i+j]!=0) {
                    for (k=1; k<j; k++) {
                        sdata[i+k]=0xff;
                    }
                }
            }

            //AND result
            __syncthreads();
            result[i] = result[i] && sdata[i];
            __syncthreads();
        }

        /////////////////////////////////////////////////////////////////////
        // the code below is used for the sum reduce
        /////////////////////////////////////////////////////////////////////
 
        sdata[i] = result[i];
        __syncthreads();

	/*
	 * conservative reduction implemented by John Emmons Feb. 2014
	 * EX. if vicinity = 3 then 111100110111111 -> 100100110100101
	 */

		
	if(sdata[i] != 0 && (i == 0 || sdata[i-1] == 0)){	
		int m, n = i;
		bool flag = true;	
		while(true){
			for(m = 1; m < vicinity + 1; m++){
				if(n + m < str_len) {
					if(sdata[n + m] != 0){ continue; }
					else if(m < 2){ flag = false; break; }
					else{ break; }
				}
				else{
					if(m < 2){ flag = false; break; }
                                        else{ break; }
				}
			}
			if(flag){
				for(m -= 2; m > 0; m--)
					sdata[n + m] = 0;
					n += vicinity;
			}
			else{ break; }	
		}
	}

	__syncthreads();

	// conservative reduction debugging
	//dev_test_array[i] = sdata[i];

        // do reduction in shared mem
        for(unsigned int s=str_len/2; s>0; s >>= 1){
            if(i < s) {
                sdata[i] += sdata[i + s];
            }
            __syncthreads();
        }
 
        __syncthreads();

        // write result for this block to global mem
        if(i%str_len == 0) {
            reduce[tid/str_len] = (sdata[i]<=pop_thresh)?0:1;
        }

        ///////////////////////////////////////////////////////////////////////

        __syncthreads();
        tid += tstride; //increase tid by thread stride amount
    }
}


/*********************************/
/** THIS FUNCTION IS DEPRECATED **/
/*********************************/

__global__ void reduce(char *g_idata, char *g_odata, unsigned long nthreads, unsigned int str_len, unsigned int pop_thresh, unsigned int tstride) {
    extern __shared__ char sdata[];
    // each thread loads one element from global to shared mem
    unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned long i = x + y * blockDim.x * gridDim.x; //global tid

    unsigned int tid = threadIdx.x; //local block tid

    while(i < nthreads) {
        sdata[tid] = g_idata[i];
        __syncthreads();
        // do reduction in shared mem
        for(unsigned int s=str_len/2; s>0; s >>= 1){
            if (tid<s) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        // write result for this block to global mem
        if (tid%str_len == 0) {
            //g_odata[i/str_len] = (sdata[tid]<=pop_thresh)?1:0;
            g_odata[i/str_len] = sdata[tid];
        }

        i += tstride;
    }
}


/*
 * Function - main
 *
 * Arguments:
 *      argc - the number of command line arguments
 *      argv - an array of the command line arguments
 *
 * Outputs:
 *      int - 0 if success, 1 if failure
 *
 * Description:
 *      This is the main function. It initializes memory. It reads in the
 *      files which contain the reads. It, then, launches the kernel on
 *      the GPU.
 */

int main(int argc, char *argv[]) {

    /* check the number of command line arguments */
    if(argc != 8) {
        usage();
        return 1;
    }

    /* get arguments */
    char* file_1;
    char* file_2;
    unsigned int num_genomes, genome_len, buffed_len, 
                 buffer_len, vicinity, errors, pop_thresh, shift_amt;
    FILE *pop_count_file;
    file_1 = argv[1]; //contains reads
    file_2 = argv[2]; //contains reads
    num_genomes = atoi(argv[3]); //the number of reads in each file
    genome_len = atoi(argv[4]); //the length of each read
    errors = atoi(argv[5]); //the number of edits allowed between two reads
    vicinity = atoi(argv[6]); //the vicinity for bit flips
    shift_amt = atoi(argv[7]); //the maximum shift amount when comparing reads

    /* calculate important values */
    pop_thresh = (vicinity-1)*(errors-1) + errors; //popcount threshold
    buffed_len = next_power_2(genome_len); //genome length + buffer space
    buffer_len = buffed_len - genome_len; //difference bw genome len and buf len
    unsigned long num_chars = num_genomes*buffed_len; //the total number of chars in every buffed read

    /* initialize and allocate strings to compare */
    char* genome_1_data; //first genome data
    char* genome_2_data; //second genome data
    char* reduce_data;   //sum of "errors" in each string
    if(init_data_pad(&genome_1_data, num_chars))  return 1;
    if(init_data_pad(&genome_2_data, num_chars))  return 1;
    if(init_data(&reduce_data, num_genomes)) return 1;

    /* conservative reduction debugging
    char* test_array;
    if(init_data(&test_array, 16)) return 1;
    */

    /* read in the data */
    if(read_data(genome_1_data, file_1, num_genomes, genome_len, buffer_len, buffed_len, vicinity+1))  return 1;
    if(read_data(genome_2_data, file_2, num_genomes, genome_len, buffer_len, buffed_len, vicinity+1))  return 1;

    /* create timing events */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* initialize and allocate memoer for GPU input and output arrays */
    char *dev_genome_1_data;
    char *dev_genome_2_data;
    char *dev_reduce_data;
    gpuErrchk( hipMalloc((void**)&dev_genome_1_data, (num_chars+PAD_AMT)*sizeof(char)));
    gpuErrchk( hipMalloc((void**)&dev_genome_2_data, (num_chars+PAD_AMT)*sizeof(char)));
    gpuErrchk( hipMalloc((void**)&dev_reduce_data, num_genomes*sizeof(char) ));

    /* conservative reduction debugging
    char *dev_test_array;
    gpuErrchk( hipMalloc((void**)&dev_test_array, 16*sizeof(char) ));
    */

      /******************/
     /** START TIMING **/
    /******************/

    /*========================================================================*/

    /* set start time */
    hipEventRecord(start, 0);

    /* copy data to GPU */    
    gpuErrchk(hipMemcpy( dev_genome_1_data, genome_1_data, 
        (num_chars+PAD_AMT)*sizeof(char), hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( dev_genome_2_data, genome_2_data, 
        (num_chars+PAD_AMT)*sizeof(char), hipMemcpyHostToDevice ));    

    /* conservative reduction debugging
    gpuErrchk(hipMemcpy( dev_test_array, test_array,
        16*sizeof(char), hipMemcpyHostToDevice ));
    */

    /* figure out thread count and dimensions for GPU */
    unsigned int num_blocks_x = 128;
    unsigned int num_blocks_y = 128;
    unsigned int threads_per_block = buffed_len;
    unsigned int tstride = threads_per_block*num_blocks_x*num_blocks_y;
    dim3 grid_size(num_blocks_x, num_blocks_y, 1);
    unsigned int log_len = log_2(buffed_len); //TODO: ALL OF THIS SHOULD PROBABLY BE MOVED ABOVE THE BEGINNING OF TIMING

    /* create and run GPU threads */
    readcmp<<<grid_size,threads_per_block,2*threads_per_block>>>(/*dev_test_array,*/ dev_genome_1_data,
            dev_genome_2_data,/* dev_result_data,*/ num_chars, buffed_len, vicinity,
            tstride, dev_reduce_data, pop_thresh, threads_per_block, shift_amt);
    gpuErrchk(hipDeviceSynchronize());

    /* write the results back */
    gpuErrchk(hipMemcpy( reduce_data, dev_reduce_data, 
                num_genomes*sizeof(char), hipMemcpyDeviceToHost ));
    
    /* conservative reduction debugging
    gpuErrchk(hipMemcpy( test_array, dev_test_array,
                16*sizeof(char), hipMemcpyDeviceToHost ));
    */

    /*========================================================================*/

    /****************/
    /** END TIMING **/
    /****************/

    /* set stop time */
    hipEventRecord(stop,0);
    hipEventSynchronize( stop );

    /* calculate elapsed time for GPU computation */
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to complete comparison %1.4f ms\n", elapsedTime);

    // Writing output pop count to file
    // for data collection purposes
    unsigned int matches=0;
    for (unsigned int q=0; q<num_genomes; q++) {
        if (reduce_data[q]==0)
            matches++;
            
    }

    /* conservative reduction debugging
    for(unsigned int i=0; i < 16; i++){
	printf("the test_array: %u\n", test_array[i]);
    }
    */

    pop_count_file = fopen("pop_output.txt","w");
    fprintf(pop_count_file, "%d %d\n", matches, num_genomes-matches);
    fclose(pop_count_file);
    
    /* free and destroy all allocated information */
    hipFree(dev_genome_1_data);
    hipFree(dev_genome_2_data);
    hipFree(dev_reduce_data);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipHostFree(genome_1_data);
    hipHostFree(genome_2_data);
    hipHostFree(reduce_data);
}

/*
 * Function - usage
 *
 * Description:
 *      Just prints the usage invariant for this program.
 */
void usage(void) {
    printf("\nUsage:\n");
    printf("\t./a.out <file_1> <file_2> <num_genomes> <genome_len> <errors> <vicinity> <adj_errs>\n\n");
}


/*
 * Function - init_data
 *
 * Arguments:
 *      data - the array in which to place data
 *      num_chars - the number of chars to allocate
 *
 * Outputs:
 *      int - 0 if success, 1 if failure
 *
 * Description:
 *      This function initializes a data array. Pretty simple to follow.
 */
int init_data(char** data, unsigned int num_chars) {
    
    /* allocate pointers for the genome strings */
    hipHostAlloc((void**)data,num_chars*sizeof(char),hipHostMallocDefault);
    if(NULL == *data) {
        printf("init_data - malloc failed\n");
        return 1;
    }

    return 0; //SUCCESS
}


/*
 * Function - init_data_pad
 *
 * Arguments:
 *      data - the array in which to place data
 *      num_chars - the number of chars to allocate
 *
 * Outputs:
 *      int - 0 if success, 1 if failure
 *
 * Description:
 *      This function initializes a data array. Pretty simple to follow. It's
 *      the same as the above function except that it adds the PAD_AMT to
 *      to it.
 */
int init_data_pad(char** data, unsigned int num_chars) {
    
    /* allocate pointers for the genome strings */
    hipHostAlloc((void**)data,(num_chars+PAD_AMT)*sizeof(char),hipHostMallocDefault);
    if(NULL == *data) {
        printf("init_data - malloc failed\n");
        return 1;
    }

    return 0; //SUCCESS
}


/*
 * Function - read_data
 *
 * Arguments:
 *      data - the char* to which the data will be written 
 *      file - the filae that contains the genomes we care about
 *      num_genomes - the numebr of genomed to read 
 *      genome_len - the length of the genomes
 *      buffer_len - the length of the buffer at the end of each genome
 *      buffed_len - length of genome + buffer
 *
 * Outputs:
 *      int - 0 if success, 1 if failure
 *
 * Description:
 *      This function reads in all of the genome data from the given genome
 *      file. Each line contains a genome read, and this is read into each
 *      string.
 */
int read_data(char* data, char* file, unsigned int num_genomes, 
                unsigned int genome_len, unsigned int buffer_len,
                unsigned int buffed_len,unsigned int vicinity) {

    /* basic info and variables */
    FILE* ifp; //ifp: "in file pointer"
    char* mode = "r";

    /* open the file */
    ifp = fopen(file, mode);
    if(NULL == ifp) {
        printf("Can't open input file %s!\n", file);
        return 1;
    }

    /* create read in buffer */
    char* buf = (char*)malloc((genome_len+2)*sizeof(char));

    /* initialize the padding at beginning of array */
    for(int i = 0; i < PAD_AMT; i++) {
        *(data + i) = BUFFER_CHAR;
    }
    int limit_len;

    /* calculate the limit to which we will read data */
    if (buffer_len < vicinity)
        limit_len = buffed_len - vicinity;
    else
        limit_len = genome_len;    

    /* read in the file */
    for(int i = 0; i < num_genomes; i++) {

        if(NULL != fgets(buf, genome_len + 2, ifp)) {
            
            for(int j = 0; j < limit_len; j++) {
                *(data + PAD_AMT + i*buffed_len + j) = buf[j];
            }

            for(int j = 0; j < buffed_len-limit_len; j++) {
                *(data + PAD_AMT + i*buffed_len + limit_len + j) = BUFFER_CHAR;
            }

        }
        else {
            printf("Failed to read from the file\n");
            return 1;
        }
    }

    /* close the file */
    fclose(ifp);
    free(buf);

    return 0; //SUCCESS
}


/*
 * Function - print_device_info
 *
 * Description:
 *      Prints valuable information out regarding the CUDA-capable devices
 *      in this system.
 */
void print_device_info(void) {
    hipDeviceProp_t  prop;

    int count;
    hipGetDeviceCount( &count ); 
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( " --- General Information for device %d ---\n", i ); 
        printf( "Name: %s\n", prop.name );
        printf( "Compute capability: %d.%d\n", prop.major, prop.minor ); 
        printf( "Clock rate: %d\n", prop.clockRate );
        printf( "Device copy overlap: " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" ); 
        else
            printf( "Disabled\n" );
        printf( "Kernel execition timeout : " ); 
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" ); 
        else
            printf( "Disabled\n" );
        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n", prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n", prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
            prop.maxThreadsDim[0], prop.maxThreadsDim[1],
            prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
            prop.maxGridSize[0], prop.maxGridSize[1],
            prop.maxGridSize[2] );
        printf( "\n" );
    }
}


/*
 * Function - next_power_2
 *
 * Arguments:
 *      v - the value for which we want to find the next power of 2
 *
 * Outputs:
 *      unsigned int - the next power of 2 greater than v
 *
 * Description:
 *      This code basically rounds v up to the next highest power of 2. So if
 *      v was 2, this function would return 2. If v was 15, this function would
 *      return 16. Etcetera.
 *
 *  Source:
 *      http://graphics.stanford.edu/~seander/bithacks.html
 */
unsigned int next_power_2(unsigned int v) {
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

unsigned int log_2(unsigned int v) {
    unsigned int r=0;
    while (v >>= 1) // unroll for more speed...
    {
        r++;
    }
    return r;
}
